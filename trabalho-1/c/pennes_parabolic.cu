#include "hip/hip_runtime.h"
#include "utils.h"
#include <time.h>


void handleCudaError(hipError_t cudaERR){
    if (cudaERR!=hipSuccess){
      printf("CUDA ERROR : %s\n", hipGetErrorString(cudaERR));
    }
}

// Computes for each thread the T_np1_{i,j}.
__global__ void solver(real *T_n, real *T_np1)
{
    int j = threadIdx.x + blockIdx.x * blockDim.x;
    int i = threadIdx.y + blockIdx.y * blockDim.y;

    int j_p = (j == tamz - 1) ? (j - 1) : (j + 1);
    int j_m = (j == 0) ? (j + 1) : (j - 1);
    int i_p = (i == tamx - 1) ? (i - 1) : (i + 1);
    int i_m = i - 1;

    real k_zp = k_harm(k(j, i), k(j_p, i));
    real k_zm = k_harm(k(j, i), k(j_m, i));
    real k_xp = k_harm(k(j, i), k(j, i_p));
    real k_xm = k_harm(k(j, i), k(j, i_m));

    T_np1[j * tamx + i] = T_n[j * tamx + i] + (ht / (rho_gpu(j,i) * c_gpu(j,i) * h * h)) * (k_xp * T_n[j*tamx + i_p] - (k_xp + k_xm) * T_n[j * tamx + i] + k_xm * T_n[j * tamx + i_m] + k_zp * T_n[j_p * tamx + i] - (k_zp + k_zm) * T_n[j * tamx + i] + k_zm * T_n[j_m * tamx + i] + h * h * (omega_b_gpu(j, i, T_n[j * tamx + i]) * c_b_gpu(j,i)*(T_a - T_n[j * tamx + i]) + Q_m_gpu(j,i) + Q_r[j * tamx + i]));   
}

int main(int argc, char* argv[]){
    read_config_txt("../inout/config.txt");
    init_vars();

    printf("Discretização espacial: %lf\n", h);
    printf("Discretização temporal: %lf\n", ht);
    printf("Número de timesteps: %d\n", tamt);

    clock_t start, end;

    /*Transfer Data from Host to Device*/
    real **T_n = (real**) alloc_matrix(tamz, tamx, sizeof(real), &T_a); // initiliazes T^n with left boundary condition.
    real **T_np1 = (real**) alloc_matrix(tamz, tamx, sizeof(real), &T_a); // initiliazes T^{n+1} with left boundary condition.

    const size_t matrixSize = sizeof(real) * size_t(tamz * tamx); 

    real *Tn_device, *Tnp1_device, *Qr_device; //pointers to access data in device
    
    //allocate poiinters in device
    handleCudaError(hipMalloc((void**)&Tn_device, matrixSize));
    handleCudaError(hipMalloc((void**)&Tnp1_device, matrixSize));
    handleCudaError(hipMalloc((void**)&Qr_device, matrixSize));

    //transfer data to device
    handleCudaError(hipMemcpy(Tn_device,   T_n,    matrixSize, hipMemcpyHostToDevice));
    handleCudaError(hipMemcpy(Tnp1_device, T_np1,  matrixSize, hipMemcpyHostToDevice));
    handleCudaError(hipMemcpy(Qr_device,   Q_r,    matrixSize, hipMemcpyHostToDevice));

    dim3 blockSize(16, 16);
    dim3 gridSize((tamx + blockSize.x - 1) / blockSize.x, (tamz + blockSize.y - 1) / blockSize.y);

    start = clock();
    for(int z = 0; z < tamt; z++){
        solver<<gridSize, blockSize>>(Tn_device, Tnp1_device); //calculates a timestep
        hipDeviceSynchronize(); //sincroniza threads após realização dos cálculos;
        real* temp = Tn_device;
        Tn_device = Tnp1_device;
        Tnp1_device = temp;
    }
    end = clock();

    real elapsed_time= (double) (end - start) / CLOCKS_PER_SEC;
    printf("Tempo paralelo: \t %f.\n", elapsed_time);

    handleCudaError(hipMemcpy(T_n,     Tn_device,      matrixSize, hipMemcpyDeviceToHost));
    handleCudaError(hipMemcpy(T_np1,   Tnp1_device,    matrixSize, hipMemcpyDeviceToHost));
    handleCudaError(hipMemcpy(Q_r,     Qr_device,    matrixSize, hipMemcpyDeviceToHost));

    free_matrix((void**) T_np1);
    free_matrix((void**) T_n);
    hipFree(Tn_device);
    hipFree(Tnp1_device);
    hipFree(Qr_device);

    end_vars();
}