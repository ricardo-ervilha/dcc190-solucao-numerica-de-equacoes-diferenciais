#include "hip/hip_runtime.h"
#include "utils.h"
#include <time.h>

// kernel to solve PDE
__global__ void solver(real *T_n, real* T_np1){
    //aux variables to compute k's and indexes
    int j_p, j_m, i_p, i_m;
    real k_zp, k_zm, k_xp, k_xm;

    int j = threadIdx.x + blockIdx.x * blockDim.x;
    int i = threadIdx.y + blockIdx.y * blockDim.y;

    j_p = (j == tamz - 1) ? (j - 1) : (j + 1);
    j_m = (j == 0) ? (j + 1) : (j - 1);
    i_p = (i == tamx - 1) ? (i - 1) : (i + 1);
    i_m = i - 1;

    k_zp = k_harm(k(j, i), k(j_p, i));
    k_zm = k_harm(k(j, i), k(j_m, i));
    k_xp = k_harm(k(j, i), k(j, i_p));
    k_xm = k_harm(k(j, i), k(j, i_m));

    T_np1[j][i] = T_n[j][i] + (ht / (rho(j,i) * c(j,i) * h * h)) * (k_xp * T_n[j][i_p] - (k_xp + k_xm) * T_n[j][i] + k_xm * T_n[j][i_m] + k_zp * T_n[j_p][i] - (k_zp + k_zm) * T_n[j][i] + k_zm * T_n[j_m][i] + h * h * (omega_b(j, i, T_n[j][i]) * c_b(j,i)*(T_a - T_n[j][i]) + Q_m(j,i) + Q_r[j][i]));  
}

int main(int argc, char* argv[]){
    /*Alocating memory*/
    read_config_txt("../inout/config.txt");
    init_vars();

    /*Printing program infos.*/
    printf("Discretização espacial: %lf\n", h);
    printf("Discretização temporal: %lf\n", ht);
    printf("Número de timesteps: %d\n", tamt);

    /*Cuda Transfer Data*/

    //Alocates in host
    real **T_n = (real**) alloc_matrix(tamz, tamx, sizeof(real), &T_a); // initiliazes T^n with left boundary condition.
    real **T_np1 = (real**) alloc_matrix(tamz, tamx, sizeof(real), &T_a); // initiliazes T^{n+1} with left boundary condition.
    real *T_ndevice, *T_np1device;
    
    const size_t sizeMatrix = sizeof(real) * size_t(tamz * tamx);
    
    //Allocates in device
    hipMalloc((void**)&T_ndevice, T_n);
    hipMalloc((void**)&T_np1device, T_np1);

    //Transfer data to device
    hipMemcpy(T_ndevice,   T_n,    sizeMatrix, hipMemcpyHostToDevice); 
    hipMemcpy(T_np1device, T_np1,  sizeMatrix, hipMemcpyHostToDevice); 

    /*Call the kernel to compute the finite difference*/
    solver<<tamz, tamx>>(T_ndevice, T_np1device);
    
    /*Desalocating memory*/
    free_matrix((void**) T_np1);
    free_matrix((void**) T_n);

    end_vars();
}